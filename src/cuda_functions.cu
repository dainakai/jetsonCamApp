#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/universal_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <tuple>

#ifdef __x86_64__
extern "C" std::tuple<float, float, float> calculateMeanStdDevK(const uint8_t* data, int size) {
    thrust::device_vector<uint8_t> d_data(size);
    thrust::copy(data, data + size, d_data.begin());

    float mean = static_cast<float>(thrust::reduce(d_data.begin(), d_data.end(), 0, thrust::plus<int>())) / size;

    thrust::device_vector<float> d_diff(size);
    thrust::transform(d_data.begin(), d_data.end(), d_diff.begin(),
                      [mean] __device__ (uint8_t val) { return (static_cast<float>(val) - mean) * (static_cast<float>(val) - mean); });

    float variance = thrust::reduce(d_diff.begin(), d_diff.end(), 0.0f, thrust::plus<float>()) / size;
    float stdDev = sqrtf(variance);

    float k = stdDev / mean;

    return std::make_tuple(mean, stdDev, k);
}
#elif __aarch64__
extern "C" std::tuple<float, float, float> calculateMeanStdDevK(uint8_t* data, int size) {
    thrust::universal_vector<uint8_t> d_data(size);
    thrust::copy(data, data + size, d_data.begin());

    float mean = static_cast<float>(thrust::reduce(d_data.begin(), d_data.end(), 0, thrust::plus<int>())) / size;

    thrust::device_vector<float> d_diff(size);
    thrust::transform(d_data.begin(), d_data.end(), d_diff.begin(),
                      [mean] __device__ (uint8_t val) { return (static_cast<float>(val) - mean) * (static_cast<float>(val) - mean); });

    float variance = thrust::reduce(d_diff.begin(), d_diff.end(), 0.0f, thrust::plus<float>()) / size;
    float stdDev = sqrtf(variance);

    float k = stdDev / mean;

    return std::make_tuple(mean, stdDev, k);
}
#else
#error "Unsupported architecture"
#endif